#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <assert.h>
#include <stdbool.h>

#define PERIOD (UINT64_C(1) << UINT64_C(42)) // h2

#define CHUNKS (1 << 19)
#define THREADS_PER_BLOCK 32

#include "generated.inc"

__global__ void test(uint64_t *result) {
    const uint64_t chunk_size = PERIOD / CHUNKS;
    assert(chunk_size * CHUNKS == PERIOD);
    int chunk = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    uint64_t magic = chunk_size * chunk;

    result[chunk] = 0;

    while (magic < chunk_size * (chunk + 1)) {
        if (check_magic(magic)) {
            result[chunk] = magic;
            return;
        }
        magic++;
    }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main() {
    printf("parallel search ...\n");

    uint64_t h_result[CHUNKS] = { 0 };
    uint64_t *d_result;
    gpuErrchk(hipMalloc(&d_result, sizeof(uint64_t) * CHUNKS));
    test<<<CHUNKS / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_result);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(h_result, d_result, sizeof(uint64_t) * CHUNKS, hipMemcpyDeviceToHost));

    printf("search complete.\n");

    for (int i = 0; i < CHUNKS; i++) {
        if (h_result[i]) printf("magic: 0x%lx\n", h_result[i]);
    }
}
